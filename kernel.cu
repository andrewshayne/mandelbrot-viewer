#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <SFML/Graphics.hpp>

#include <stdio.h>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <string>

using namespace std;

//__device__ __constant__ int THREAD_COUNT = 100;


int mandelbrot_type = 1;
int MAX_ITERATIONS = 1000;
int LINES = 500;
const int WIDTH = 1024;
const int HEIGHT = 1024;

__device__ __constant__ int const_WIDTH = WIDTH;
__device__ __constant__ int const_HEIGHT = HEIGHT;
//sf::Uint8* buff;
uint8_t* buff;

//iteration buffer
int* iteration_buff;


long double x_lower = -1.5;
long double x_upper = 1.5;
long double y_lower = -1.5;
long double y_upper = 1.5;

//set point to zoom in on and window size around it

// origin
long double shrink_factor = 1.5;
long double x_origin = 0.0;
long double y_origin = 0.0;

// full view of Z^2
//long double shrink_factor = 1.5;
//long double x_origin = -0.75;
//long double y_origin = 0.0;

// cool spiral coords!
//long double shrink_factor = 0.000046;
//long double x_origin = -0.530859;
//long double y_origin = -0.592524;



//consts for row coloring function below...
__device__ __constant__ int const_rgb_1[3];
__device__ __constant__ int const_rgb_2[3];
__device__ __constant__ int const_rgb_diff[3];

int rgb_1[3] = { 124, 0, 181 };
int rgb_2[3] = { 230, 192, 25 };
int rgb_diff[3] = { rgb_2[0] - rgb_1[0], rgb_2[1] - rgb_1[1], rgb_2[2] - rgb_1[2] };


//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t setBuffWithCuda(uint8_t* buff, int* iteration_buff, const long double x_lower, const long double x_upper, const long double y_lower, const long double y_upper, const int MAX_ITERATIONS, const int mandelbrot_type);

__global__ void setPixelKernel(uint8_t* buff, int* iteration_buff, const long double x_lower, const long double x_upper, const long double y_lower, const long double y_upper, const int MAX_ITERATIONS, const int mandelbrot_type)
{
	//derive i and j from index...
	int i = blockIdx.x * blockDim.x + threadIdx.x;// index;
	int j = blockIdx.y * blockDim.y + threadIdx.y;// index;


	long double x = x_lower + i / (const_WIDTH - 1.0)*(x_upper - x_lower);
	long double y = y_lower + j / (const_HEIGHT - 1.0)*(y_upper - y_lower);
	//complex Z(0.0, 0.0);
	//complex C(x, y);

	long double Z_a = 0.0;
	long double Z_b = 0.0;
	long double C_a = x;
	long double C_b = y;

	int count;
	long double a;
	long double b;
	for (count = 0; count < MAX_ITERATIONS && (Z_a*Z_a + Z_b*Z_b) < 4.0; count++) {
		a = Z_a;
		b = Z_b;

		long double a2 = a * a;
		long double b2 = b * b;
		long double a4 = a2 * a2;
		long double b4 = b2 * b2;

		switch (mandelbrot_type)
		{
		case 0:
			// burning ship
			Z_a = a2 - b2 + C_a;
			Z_b = 2.0 * a * b;
			if (Z_b < 0) Z_b = -Z_b;
			Z_b += C_b;
			break;
		case 1:
			// Z = Z^2 + C;
			Z_a = (a*a - b * b) + C_a;
			Z_b = (2.0 * a * b) + C_b;
			break;

			// Mandelbar (Z = Z_bar^2 + C)
			//Z_a = (a*a - b*b) + C_a;
			//Z_b = (-2.0 * a * b) + C_b;

		case 2:
			// Z = Z^3 + C;
			Z_a = a*(a2 - 3.0*b2) + C_a;
			Z_b = b*(3.0*a2 - b2) + C_b;
			break;

		case 3:
			// Z = Z^4 + C;
			Z_a = a2*a2 -6.0*a2*b2 + b2*b2 + C_a;
			Z_b = 4.0*a*b*(a2 - b2) + C_b;
			break;

		case 5:
			// Z = Z^6 + C;
			Z_a = a4*a2 - 15.0*a4*b2 + 15.0*a2*b4 - b4*b2 + C_a;
			Z_b = 6.0*a4*a*b -20.0*a2*a*b2*b + 6*a*b4*b + C_b;
		}

	}

	iteration_buff[j*const_HEIGHT + i] = count;

	//sf::Color myColor = sf::Color(
	//	(const_rgb_diff[0] * count / M) + const_rgb_1[0],
	//	(const_rgb_diff[1] * count / M) + const_rgb_1[1],
	//	(const_rgb_diff[2] * count / M) + const_rgb_1[2]
	//);


	//if (count != MAX_ITERATIONS) {
	//	buff[4 * (j*const_HEIGHT + i)]	   = (const_rgb_diff[0] * count / MAX_ITERATIONS) + const_rgb_1[0];
	//	buff[4 * (j*const_HEIGHT + i) + 1] = (const_rgb_diff[1] * count / MAX_ITERATIONS) + const_rgb_1[1];
	//	buff[4 * (j*const_HEIGHT + i) + 2] = (const_rgb_diff[2] * count / MAX_ITERATIONS) + const_rgb_1[2];
	//	buff[4 * (j*const_HEIGHT + i) + 3] = 255;
	//}

	if (count != MAX_ITERATIONS) {
		float H = (count+360) % 360;
		float S = 0.5;
		float V = 0.75;
		float C = V * S;

		float partAbs = fmod((double)(H / 60.0), 2.0) - 1.0;
		partAbs = (partAbs > 0 ? partAbs : -partAbs);
		float X = C * (1 - partAbs);
		float m = V - C;

		float Rp = 0;
		float Gp = 0;
		float Bp = 0;
		//int R; int G; int B;

		if (H < 60) {
			Rp = C; Gp = X; Bp = 0;
		} else if (H < 120) {
			Rp = X; Gp = C; Bp = 0;
		} else if (H < 180) {
			Rp = 0; Gp = C; Bp = X;
		} else if (H < 240) {
			Rp = 0; Gp = X; Bp = C;
		} else if (H < 300) {
			Rp = X; Gp = 0; Bp = C;
		} else { // H < 360
			Rp = C; Gp = 0; Bp = X;
		}

		buff[4 * (j*const_HEIGHT + i)]	   = (Rp+m)*255;
		buff[4 * (j*const_HEIGHT + i) + 1] = (Gp+m)*255;
		buff[4 * (j*const_HEIGHT + i) + 2] = (Bp+m)*255;
		buff[4 * (j*const_HEIGHT + i) + 3] = 255;
	}
	else {
		buff[4 * (j*const_HEIGHT + i)]     = 0;
		buff[4 * (j*const_HEIGHT + i) + 1] = 0;
		buff[4 * (j*const_HEIGHT + i) + 2] = 0;
		buff[4 * (j*const_HEIGHT + i) + 3] = 0;
	}
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void cpuColorPixel(int count, int i, int j, int color_tastes_like_honey)
{
	if (count != MAX_ITERATIONS) {
		count += color_tastes_like_honey;
		float H = (count+360) % 360;
		float S = 0.5;
		float V = 0.75;
		float C = V * S;

		float partAbs = fmod((double)(H / 60.0), 2.0) - 1.0;
		partAbs = (partAbs > 0 ? partAbs : -partAbs);
		float X = C * (1 - partAbs);
		float m = V - C;

		float Rp = 0;
		float Gp = 0;
		float Bp = 0;
		//int R; int G; int B;

		if (H < 60) {
			Rp = C; Gp = X; Bp = 0;
		} else if (H < 120) {
			Rp = X; Gp = C; Bp = 0;
		} else if (H < 180) {
			Rp = 0; Gp = C; Bp = X;
		} else if (H < 240) {
			Rp = 0; Gp = X; Bp = C;
		} else if (H < 300) {
			Rp = X; Gp = 0; Bp = C;
		} else { // H < 360
			Rp = C; Gp = 0; Bp = X;
		}


		buff[4 * (j*HEIGHT + i)]	   = (Rp+m)*255;
		buff[4 * (j*HEIGHT + i) + 1] = (Gp+m)*255;
		buff[4 * (j*HEIGHT + i) + 2] = (Bp+m)*255;
		buff[4 * (j*HEIGHT + i) + 3] = 255;
	}
	else {
		buff[4 * (j*HEIGHT + i)]     = 0;
		buff[4 * (j*HEIGHT + i) + 1] = 0;
		buff[4 * (j*HEIGHT + i) + 2] = 0;
		buff[4 * (j*HEIGHT + i) + 3] = 0;
	}
}

// Helper function for using CUDA to add vectors in parallel. - MODIFY FOR MANDELBROT
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
hipError_t setBuffWithCuda(uint8_t* buff, int* iteration_buff,
	const long double x_lower, const long double x_upper, const long double y_lower, const long double y_upper,
	const int MAX_ITERATIONS, const int mandelbrot_type)
{
	uint8_t* dev_buff;
	int* dev_iteration_buff;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	// * Allocate for mandelbrot screen buffer
	unsigned int size = WIDTH * HEIGHT;
    cudaStatus = hipMalloc((void**)&dev_buff, size * 4 * sizeof(uint8_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// * Allocate for mandelbrot screen iteration buffer
	cudaStatus = hipMalloc((void**)&dev_iteration_buff, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


	// NOT NECESSARY- every computation is completely fresh
	/*
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	*/

    // Launch a kernel on the GPU with one thread for each element.
	dim3 blocksPerGrid(32, 32, 1);
	dim3 threadsPerBlock(32, 32, 1);
	setPixelKernel <<<blocksPerGrid, threadsPerBlock>>> (dev_buff, dev_iteration_buff, x_lower, x_upper, y_lower, y_upper, MAX_ITERATIONS, mandelbrot_type);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(buff, dev_buff, size * 4 * sizeof(uint8_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(iteration_buff, dev_iteration_buff, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_buff);
    hipFree(dev_iteration_buff);
    
    return cudaStatus;
}




int main()
{
	int color_mod = 0;
	int color_mod_incr = 0;

	bool hideText = false;
	bool drawLines = false;
	bool refreshIterations = false;

	hipMemcpyToSymbol(HIP_SYMBOL(const_rgb_1), rgb_1, sizeof(rgb_1));
	hipMemcpyToSymbol(HIP_SYMBOL(const_rgb_2), rgb_2, sizeof(rgb_2));
	hipMemcpyToSymbol(HIP_SYMBOL(const_rgb_diff), rgb_diff, sizeof(rgb_diff));

	sf::Clock clock;
	std::setprecision(10);

	//x_origin = -0.59990625;
	//y_origin = -0.4290703125;
	//shrink_factor = 0.001;

	x_lower = x_origin - shrink_factor;
	x_upper = x_origin + shrink_factor;
	y_lower = y_origin - shrink_factor;
	y_upper = y_origin + shrink_factor;

	long double x_interval = (x_upper - x_lower) / (long double)WIDTH;
	long double y_interval = (y_upper - y_lower) / (long double)HEIGHT;
	sf::RenderWindow window(sf::VideoMode(WIDTH, HEIGHT), "Mandelbrot Viewer");

	//buff = new sf::Uint8[WIDTH * HEIGHT * 4];
	buff = new uint8_t[WIDTH * HEIGHT * 4];
	iteration_buff = new int[WIDTH * HEIGHT];
	sf::Image buf;
	//buf.create(WIDTH, HEIGHT, sf::Color::Black);

	sf::Sprite mSprite;
	sf::Texture mTexture;

	sf::Font font;
	font.loadFromFile("Resources/BELLB.TTF");

	//TEXT
	sf::Text centerText;
	centerText.setFont(font);
	centerText.setPosition({10.f,0.f});

	sf::Text rectText;
	rectText.setFont(font);
	rectText.setPosition({10.f,30.f});

	sf::Text colorRotateText;
	colorRotateText.setFont(font);
	colorRotateText.setPosition({10.f,90.f});

	sf::Text iterationText;
	iterationText.setFont(font);
	iterationText.setPosition({10.f,150.f});

	//controls...
	sf::Text lineText;
	lineText.setFont(font);
	lineText.setPosition({10.f,210.f});

	//hide text
	sf::Text hideTextText;
	hideTextText.setFont(font);
	hideTextText.setPosition({740.f,0.f});


	centerText.setString("Center: " + to_string(x_origin) + ", " + to_string(y_origin));
	rectText.setString("ViewPort Size: " + to_string(shrink_factor));
	colorRotateText.setString("Color Rotation Speed (\'A\'/\'B\'): 0");
	iterationText.setString("Iterations (\'Up\'/\'Down\'): " + to_string(MAX_ITERATIONS));
	lineText.setString("Toggle Line Mode: \'L\'");
	hideTextText.setString("Press H to hide text");

	setBuffWithCuda(buff, iteration_buff, x_lower, x_upper, y_lower, y_upper, MAX_ITERATIONS, mandelbrot_type);
	buf.create(WIDTH, HEIGHT, (const uint8_t*)buff);
	mTexture.loadFromImage(buf);
	mSprite.setTexture(mTexture);


	//if I click on the screen, make it "shrink" the rectangle window by factor of 10 around that point!

	bool currentlyRendering = false;

	while (window.isOpen())
	{
		sf::Event event;
		while (window.pollEvent(event))
		{
			if (event.type == sf::Event::Closed)
				window.close();

			bool changedMandelbrotType = false;

			//key pressed
			if (event.type == sf::Event::KeyPressed && !currentlyRendering)
			{
				//toggle text display
				if (event.key.code == sf::Keyboard::H)
				{
					hideText = !hideText;
				}

				//alter color rotation speed
				if (event.key.code == sf::Keyboard::A)
				{
					color_mod_incr++;
					printf("increase color mod: %i\n", color_mod_incr);
					colorRotateText.setString("Color Rotation Speed: " + to_string(color_mod_incr));
				}
				if (event.key.code == sf::Keyboard::B)
				{
					color_mod_incr--;
					//if (color_mod_incr < 0) color_mod_incr = 0;
					printf("decrease color mod: %i\n", color_mod_incr);
					colorRotateText.setString("Color Rotation Speed: " + to_string(color_mod_incr));
				}

				//double/half iteration count
				//ooooooor +/- 1000
				if (event.key.code == sf::Keyboard::Up)
				{
					MAX_ITERATIONS += 1000;
					iterationText.setString("Iterations: " + to_string(MAX_ITERATIONS));
					refreshIterations = true;
				}
				if (event.key.code == sf::Keyboard::Down)
				{
					MAX_ITERATIONS -= 1000;
					if (MAX_ITERATIONS <= 0)
						MAX_ITERATIONS = 2;
					iterationText.setString("Iterations: " + to_string(MAX_ITERATIONS));
					refreshIterations = true;
				}

				//toggle line render
				if (event.key.code == sf::Keyboard::L)
				{
					drawLines = !drawLines;
					//lineText.setString("Toggle Lines:")
				}

				if (event.key.code == sf::Keyboard::Num0)
				{
					mandelbrot_type = 0;
					changedMandelbrotType = true;
				}
				if (event.key.code == sf::Keyboard::Num1)
				{
					mandelbrot_type = 1;
					changedMandelbrotType = true;
				}
				if (event.key.code == sf::Keyboard::Num2)
				{
					mandelbrot_type = 2;
					changedMandelbrotType = true;
				}
				if (event.key.code == sf::Keyboard::Num3)
				{
					mandelbrot_type = 3;
					changedMandelbrotType = true;
				}
				if (event.key.code == sf::Keyboard::Num5)
				{
					mandelbrot_type = 5;
					changedMandelbrotType = true;
				}

			}

			//mouse pressed
			if (((event.type == sf::Event::MouseButtonPressed && (event.mouseButton.button == sf::Mouse::Left || event.mouseButton.button == sf::Mouse::Right)) || event.type == sf::Event::MouseWheelScrolled || refreshIterations || changedMandelbrotType || changedMandelbrotType) && !currentlyRendering) {
				clock.restart();

				//set origin and reduce shrink!

				//scroll
				if (event.type == sf::Event::MouseWheelScrolled)
				{
					if (event.mouseWheelScroll.delta > 0)
						shrink_factor *= 0.5;
					else if (event.mouseWheelScroll.delta < 0)
						shrink_factor *= 2.0;
				}

				//click
				if (event.type == sf::Event::MouseButtonPressed)
				{
					if (event.mouseButton.button == sf::Mouse::Left)
						shrink_factor *= 0.66666666667;
					else if (event.mouseButton.button == sf::Mouse::Right)
						shrink_factor *= 1.5;
				}

				//do not recalculate transformation if we are only refreshing
				if (!refreshIterations)
				{
					//get mouse pixel coords
					int x_window = sf::Mouse::getPosition(window).x;
					int y_window = sf::Mouse::getPosition(window).y;
	
					x_origin = (x_upper - x_lower)*x_window / WIDTH + x_lower;
					y_origin = (y_upper - y_lower)*y_window / HEIGHT + y_lower;

					x_lower = x_origin - shrink_factor;
					x_upper = x_origin + shrink_factor;
					y_lower = y_origin - shrink_factor;
					y_upper = y_origin + shrink_factor;

					//set origin to 0 if changed type
					if (changedMandelbrotType)
					{
						x_origin = 0;
						y_origin = 0;
						shrink_factor = 1.5f;

						x_lower = x_origin - shrink_factor;
						x_upper = x_origin + shrink_factor;
						y_lower = y_origin - shrink_factor;
						y_upper = y_origin + shrink_factor;
					}

					centerText.setString("Center: " + to_string(x_origin) + ", " + to_string(y_origin));
					rectText.setString("ViewPort Size: " + to_string(shrink_factor));
				}

				currentlyRendering = true;

				setBuffWithCuda(buff, iteration_buff, x_lower, x_upper, y_lower, y_upper, MAX_ITERATIONS, mandelbrot_type);

				// uint8_t alias to sf::Uint8
				buf.create(WIDTH, HEIGHT, (const sf::Uint8*)buff);
				//buf.create(WIDTH, HEIGHT, (const uint8_t*)buff);
				mTexture.loadFromImage(buf);
				mSprite.setTexture(mTexture); //code gets OOF'd right here when iterations get too high
				currentlyRendering = false;

				sf::Time renderTime = clock.getElapsedTime();
				printf("time to render: %f\n", renderTime.asSeconds());

				refreshIterations = false;
			}

		}

		color_mod += color_mod_incr;
		if (color_mod > 360)
		{
			color_mod = 0;
		}
		else if (color_mod < 0)
		{
			color_mod = 360;
		}

		//color rotate each pixel
		if (color_mod_incr != 0)
		{
			for (int i = 0; i < HEIGHT; i++)
			{
				for (int j = 0; j < WIDTH; j++)
				{
					cpuColorPixel(iteration_buff[j*HEIGHT + i], i, j, color_mod);
				}
			}
		}

		//set trippy stuff for rendering
		buf.create(WIDTH, HEIGHT, (const sf::Uint8*)buff);
		mTexture.loadFromImage(buf);
		mSprite.setTexture(mTexture);

		window.clear();
		window.draw(mSprite);

		//LINES
		if (drawLines)
		{

			//get mouse pixel coords
			float x_window = sf::Mouse::getPosition(window).x;
			float y_window = sf::Mouse::getPosition(window).y;

			float x = (x_upper - x_lower)*x_window / WIDTH + x_lower;
			float y = (y_upper - y_lower)*y_window / HEIGHT + y_lower;

			float Z_a = 0.0;
			float Z_b = 0.0;
			float C_a = x;
			float C_b = y;

			float a;
			float b;

			sf::Vertex* line = new sf::Vertex[LINES*2];

			for (int i = 0; i < LINES; i++)
			{
				a = Z_a;
				b = Z_b;

				long double a2 = a * a;
				long double b2 = b * b;
				long double a4 = a2 * a2;
				long double b4 = b2 * b2;

				switch (mandelbrot_type)
				{
				case 0:
					// burning ship
					Z_a = a2 - b2 + C_a;
					Z_b = 2.0 * a * b;
					if (Z_b < 0) Z_b = -Z_b;
					Z_b += C_b;
					break;
				case 1:
					// Z = Z^2 + C;
					Z_a = (a*a - b * b) + C_a;
					Z_b = (2.0 * a * b) + C_b;
					break;

					// Mandelbar (Z = Z_bar^2 + C)
					//Z_a = (a*a - b*b) + C_a;
					//Z_b = (-2.0 * a * b) + C_b;

				case 2:
					// Z = Z^3 + C;
					Z_a = a * (a2 - 3.0*b2) + C_a;
					Z_b = b * (3.0*a2 - b2) + C_b;
					break;

				case 3:
					// Z = Z^4 + C;
					Z_a = a2 * a2 - 6.0*a2*b2 + b2 * b2 + C_a;
					Z_b = 4.0*a*b*(a2 - b2) + C_b;
					break;

				case 5:
					// Z = Z^6 + C;
					Z_a = a4 * a2 - 15.0*a4*b2 + 15.0*a2*b4 - b4 * b2 + C_a;
					Z_b = 6.0*a4*a*b - 20.0*a2*a*b2*b + 6 * a*b4*b + C_b;
				}

				line[i * 2] = sf::Vertex({ x_window, y_window });

				x_window = (a - x_lower)*WIDTH / (x_upper - x_lower);
				y_window = (b - y_lower)*HEIGHT / (y_upper - y_lower);

				line[i * 2 + 1] = sf::Vertex({ x_window, y_window });
			}
			window.draw(line, LINES, sf::Lines);
		}

		//TEXT
		if (!hideText)
		{
			window.draw(centerText);
			window.draw(rectText);
			window.draw(colorRotateText);
			window.draw(iterationText);
			window.draw(lineText);
			window.draw(hideTextText);
		}

		//DISPLAY
		window.display();
	}

	return 0;
}


